#include "hip/hip_runtime.h"
#include <SiftFeatures.h>




__global__ void Convolution(float* image,float* mask, ArrayImage* PyDoG, int maskR,int maskC, int imgR,int imgC, float* imgOut, int idxPyDoG){
	int tid= threadIdx.x;
	int bid= blockIdx.x;
	int bDim=blockDim.x;
	int gDim=gridDim.x;
	
		
	int iImg=0;
	float aux=0;
	int pxlThrd = ceil((double)(imgC*imgR)/(gDim*bDim)); ////////numero de veces que caben
														 ////////los hilos en la imagen.
	for(int i = 0; i <pxlThrd; ++i)///////////////////////////// Strike 
	{
		//////////////////////////////////////
		//////////////////////////////////////Calculo de indices
		iImg=(tid+(bDim*bid)) + (i*gDim*bDim); //// pixel en el que trabajara el hilo
		//////////////////////////////////////
		//////////////////////////////////////

		if(iImg < imgC*imgR){
			int condition=maskC/2+imgC*(floor((double)maskC/2));
			if (iImg-condition < 0  ||										///condicion arriba
				iImg+condition > imgC*imgR ||								///condicion abajo
				iImg%imgC < maskC/2 ||										///condicion izquierda
				iImg%imgC > (imgC-1)-(maskC/2) )							///condicion derecha
			{
				aux=0;;
			}else{		
				int itMask = 0;
				int itImg=iImg-condition;
				for (int j = 0; j < maskR; ++j)
				{		
					for (int h = 0; h < maskC; ++h)
					{
						aux+=image[itImg]*mask[itMask];
						++itMask;
						++itImg;
					}
					itImg+=imgC-maskC;
				}
			}
			//aux=(aux<0)?0:aux;
			imgOut[iImg]=aux;//(aux>255)?255:aux;
			aux=0;
		}
	}
	PyDoG[idxPyDoG].image=imgOut;
}

////////////////////arreglar   
__global__ void LocateMaxMin(ArrayImage* PyDoG, int idxPyDoG , float * imgOut ,int maskC, int imgR,int imgC)
{
	int tid= threadIdx.x;
	int bid= blockIdx.x;
	int bDim=blockDim.x;
	int gDim=gridDim.x;
	
		
	int iImg=0;
	int pxlThrd = ceil((double)(imgC*imgR)/(gDim*bDim)); ////////numero de veces que caben
														 ////////los hilos en la imagen.

	for(int i = 0; i <pxlThrd; ++i)///////////////////////////// Strike 
	
	{
		int min=0;
		int max=0;
		float value=0.0;
		float compare =0.0;
		//////////////////////////////////////
		//////////////////////////////////////Calculo de indices
		iImg=(tid+(bDim*bid)) + (i*gDim*bDim); //// pixel en el que trabajara el hilo
		//////////////////////////////////////
		//////////////////////////////////////
		
		if(iImg < imgC*imgR){
			
			
			int condition=(maskC/2)+imgC*(maskC/2);
			if (iImg-condition < 0  ||										///condicion arriba
				iImg+condition > imgC*imgR ||								///condicion abajo
				iImg%imgC < maskC/2 ||										///condicion izquierda
				iImg%imgC > (imgC-1)-(maskC/2) )							///condicion derecha
			{
				imgOut[iImg]=0.5;				
			}
			else{
				imgOut[iImg]=1.0;
				value=PyDoG[idxPyDoG].image[iImg];
				
				for (int m = -1; m < 2; ++m)
				{
					int itImg=iImg-(1+imgC);
					
					for (int j = 0; j < 3; ++j)
					{		
						for (int h = 0; h < 3; ++h)
						{
							compare =PyDoG[idxPyDoG+m].image[itImg];
							//if(iImg==10000)printf("%i iImg %i itImg  %i m  %i j %i h\n",iImg,itImg, m,j,h );
							if(value<compare && max==0)
							{
								++min;
							}
							else if(value>compare && min==0)
							{
								++max;
							}
							++itImg;
						}
						itImg+=imgC-3;
					}
				}
  
				if(min==26){
					/////Es Punto extremo;
					 imgOut[iImg]=0.0;
				}else if(max==26){
					/////Es Punto extremo;
					 imgOut[iImg]=0.0;
				}else{
					imgOut[iImg]=0.5;
				}
			
            }
		}
	}
}








void MaskGenerator(double sigma, int size,Mat mask){//Generate Gaussian Kernel
	Mat aux = getGaussianKernel(size,sigma,CV_32F);
	Mat aux_t;
	transpose(aux,aux_t);
	mask=aux*aux_t;
}

int ResizeImage(Mat image,vector<Mat>& images, int octvs){
	images.push_back(image);
	for(int i=0; i<octvs-1; ++i)
	{
		Mat aux = images[i];
		resize(aux,aux,Size(images[i].cols/2,images[i].rows/2));
		images.push_back(aux);
	}
	return 0;
}

int PyramidKDoG(vector<Mat> & PyKDoG, int octvs, int intvls){
	vector<double> sig;
	double k = 1.5;
	double sigma= sqrt(2)/6;
	vector<Mat> PyGauss;
	int size = 9;//size of gaussian mask
	Mat mask=Mat::ones(size,size,CV_32F);
	MaskGenerator(sigma,size,mask);
	PyGauss.push_back(mask);

	for(int i=1; i<intvls+3; ++i){	
		Mat aux=Mat::ones(size,size,CV_32F);
		sigma*=k;
		MaskGenerator(sigma,size,aux);
		PyGauss.push_back(aux);
		
	}
	///////////////////////////////////////////////////////////////////////////////////////
	/////////////////////////////////////////////////////////////////////Resta de Gausianas
	for(int i=0; i<intvls+2; ++i){
		Mat aux=Mat::ones(size,size,CV_32F);
		subtract(PyGauss[i+1],PyGauss[i],aux);
		PyKDoG.push_back(aux);
	}
	///////////////////////////////////////////////////////////////////////////////////////
	return 0;
}

int SiftFeatures(Mat Image, vector<Mat> PyDoG){
	const int intvls = 2;
	int octvs;
	//hipError_t e;
	octvs = log( min( Image.rows, Image.cols ) ) / log(2) - 2;
	vector<Mat> PyKDoG;
	vector<Mat> images;
	PyramidKDoG( PyKDoG,octvs,intvls);
	ResizeImage(Image,images,octvs);
	int idxPyDoG=0;
	
	
	ArrayImage * pyDoG;
	//MinMax * minMax;
	int mMidx=1;
	////////////////////////////////////////////////////////////////////////////////////////
	/////////////////////////////////////////////////////////////////////Reservo Memoria GPU
	hipMalloc(&pyDoG,sizeof(ArrayImage)*images.size()*sizeof(ArrayImage) *PyKDoG.size());
	//hipMalloc(&minMax,sizeof(MinMax)/*No se tamaño del arreglo*/);
	//cout<<hipGetErrorString(e)<<" hipMalloc"<<endl;

	for (int i = 0; i < images.size() ; ++i)
	//for (int i = 0; i < 1 ; ++i)
	{
		
		float * img_D;
		int sizeImage = images[i].rows*images[i].cols;
		
		////////////////////////////////////////////////////////////////////////////////////////
		/////////////////////////////////////////////////////////////////////Reservo Memoria GPU
		
		hipMalloc(&img_D,sizeof(float)*sizeImage);///imagenes
		//cout<<hipGetErrorString(e)<<" hipMalloc"<<endl;
	
		////////////////////////////////////////////////////////////////////////////////////////
		/////////////////////////////////////////////////////////////////////Copio Memoria GPU

		hipMemcpy(img_D,images[i].ptr<float>(),sizeof(float)*sizeImage,hipMemcpyHostToDevice);
		//cout<<hipGetErrorString(e)<<" cudaMemCopyHD"<<endl;

		int imgBlocks= ceil((double) images[i].cols/BW);
		
		////////////////////////////////////////////////////////////////////////////////////////
		///////////////////////////////////////////////////////Convolucion de mascara con imagen
		/////////////////////////////////////////////////////////////////////Una Octava or ciclo
		for (int m = 0; m < PyKDoG.size(); ++m){
			float * pkDoG_D;
			float * out_D;
			float * out= new float[sizeImage];
			int sizeMask=PyKDoG[m].rows*PyKDoG[m].cols;

			////////////////////////////////////////////////////////////////////////////////////////
			/////////////////////////////////////////////////////////////////////Reservo Memoria GPU
			hipMalloc(&pkDoG_D,sizeof(float)*sizeMask);//mascaras
			//cout<<hipGetErrorString(e)<<" cudaMalloc________Mask "<<endl;
			hipMalloc(&out_D,sizeof(float)*sizeImage);
			//cout<<hipGetErrorString(e)<<" cudaMalloc________Mask"<<endl;
			////////////////////////////////////////////////////////////////////////////////////////
			/////////////////////////////////////////////////////////////////////Copio Memoria GPU

			hipMemcpy(pkDoG_D,PyKDoG[m].ptr<float>(),sizeof(float)*sizeMask,hipMemcpyHostToDevice);
			//cout<<hipGetErrorString(e)<<" cudaMemCopyHD________Mask"<<endl;

			////////////////////////////////////////////////////////////////////////////////////////
			/////////////////////////////////////////////////////////////////////Lanzo Kernel
			
			Convolution<<<imgBlocks,1024>>>(img_D,pkDoG_D,pyDoG,PyKDoG[m].rows,PyKDoG[m].cols,images[i].rows,images[i].cols,out_D,idxPyDoG);
			hipDeviceSynchronize();
			++idxPyDoG;
			hipFree(pkDoG_D); 
			

			hipMemcpy(out,out_D,sizeof(float)*sizeImage,hipMemcpyDeviceToHost);
			//cout<<hipGetErrorString(e)<<" cudaMemCopyDH________Mask"<<endl;

			Mat image_out(images[i].rows,images[i].cols,CV_32F,out);
			
			imshow("tesuto",image_out*5);
    		waitKey(0);
    		destroyAllWindows();

			delete(out);
			//hipFree(out_D);
		}
		hipFree(img_D);
		////////////////////////////////////////////////////////////////////////////////////////
		////////////////////////////////////////////////////////////////////////////////////////
		////////////////////////////////////////////////////////////////////////////////////////

		////////////////////////////////////////////////////////////////////////////////////////
		//////////////////////////////////////////////////////////////////////Busqueda de MinMax
		/////////////////////////////////////////////////////////////////////Una Octava or ciclo
		for(int m = mMidx; m < idxPyDoG-1; ++m){
			
			float * out_D;
			float * out= new float[sizeImage];
			////////////////////////////////////////////////////////////////////////////////////////
			/////////////////////////////////////////////////////////////////////Reservo Memoria GPU
			hipMalloc(&out_D,sizeof(float)*sizeImage);
			//cout<<hipGetErrorString(e)<<" cudaMalloc________Mask"<<endl;
						
			////////////////////////////////////////////////////////////////////////////////////////
			/////////////////////////////////////////////////////////////////////Lanzo Kernel
			cout<<m<<endl;
			LocateMaxMin<<<imgBlocks,1024>>>(pyDoG,m,out_D,PyKDoG[m].cols,images[i].rows,images[i].cols);
			//LocateMaxMin<<<1,1>>>(pyDoG,mMidx,out_D,PyKDoG[m].cols,images[i].rows,images[i].cols);
			hipDeviceSynchronize();
						

			hipMemcpy(out,out_D,sizeof(float)*sizeImage,hipMemcpyDeviceToHost);
			//cout<<hipGetErrorString(e)<<" cudaMemCopyDH________Mask"<<endl;

			Mat image_out(images[i].rows,images[i].cols,CV_32F,out);
			
			imshow("tesuto",image_out);
    		waitKey(0);
    		destroyAllWindows();

			delete(out);
			//hipFree(out_D);
		}
		mMidx=idxPyDoG+1;
		////////////////////////////////////////////////////////////////////////////////////////
		////////////////////////////////////////////////////////////////////////////////////////
		////////////////////////////////////////////////////////////////////////////////////////

	}
	hipFree(pyDoG);



	return 0;
}
