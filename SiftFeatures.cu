#include "hip/hip_runtime.h"
#include <SiftFeatures.h>




__global__ void Convolution(float* image,float* mask, ArrayImage* PyDoG, int maskR,int maskC, int imgR,int imgC, float* imgOut, int idxPyDoG)
{
	int tid= threadIdx.x;
	int bid= blockIdx.x;
	int bDim=blockDim.x;
	int gDim=gridDim.x;
	
		
	int iImg=0;
	float aux=0;
	int pxlThrd = ceil((double)(imgC*imgR)/(gDim*bDim)); ////////numero de veces que caben
														 ////////los hilos en la imagen.
	for(int i = 0; i <pxlThrd; ++i)///////////////////////////// Strike 
	{
		//////////////////////////////////////
		//////////////////////////////////////Calculo de indices
		iImg=(tid+(bDim*bid)) + (i*gDim*bDim); //// pixel en el que trabajara el hilo
		//////////////////////////////////////
		//////////////////////////////////////

		if(iImg < imgC*imgR){
			int condition=maskC/2+imgC*(floor((double)maskC/2));
			if (iImg-condition < 0  ||										///condicion arriba
				iImg+condition > imgC*imgR ||								///condicion abajo
				iImg%imgC < maskC/2 ||										///condicion izquierda
				iImg%imgC > (imgC-1)-(maskC/2) )							///condicion derecha
			{
				aux=0;
			}else{		
				int itMask = 0;
				int itImg=iImg-condition;
				for (int j = 0; j < maskR; ++j)
				{		
					for (int h = 0; h < maskC; ++h)
					{
						aux+=image[itImg]*mask[itMask];
						++itMask;
						++itImg;
					}
					itImg+=imgC-maskC;
				}
			}
			
			
			imgOut[iImg]=aux;
			aux=0;
		}
	}
	PyDoG[idxPyDoG].image=imgOut;
}

__global__ void LocateMaxMin(ArrayImage* PyDoG, int idxPyDoG , float * imgOut ,MinMax * mM, int maskC, int imgR,int imgC, int idxmM)
{
	int tid= threadIdx.x;
	int bid= blockIdx.x;
	int bDim=blockDim.x;
	int gDim=gridDim.x;
	
	int iImg=0;
	int pxlThrd = ceil((double)(imgC*imgR)/(gDim*bDim)); ////////numero de veces que caben
														 ////////los hilos en la imagen.

	for(int i = 0; i <pxlThrd; ++i)///////////////////////////// Strike 
	
	{
		//int min=0;
		//int max=0;
		float value=0.0;
		//float compare =0.0;
		//////////////////////////////////////
		//////////////////////////////////////Calculo de indices
		iImg=(tid+(bDim*bid)) + (i*gDim*bDim); //// pixel en el que trabajara el hilo
		//////////////////////////////////////
		//////////////////////////////////////
		
		if(iImg < imgC*imgR){
			
			
			int condition=maskC/2+imgC*(floor((double)maskC/2));
			if (iImg-condition < 0  ||										///condicion arriba
				iImg+condition > imgC*imgR ||								///condicion abajo
				iImg%imgC < maskC/2 ||										///condicion izquierda
				iImg%imgC > (imgC-1)-(maskC/2) )							///condicion derecha
			{                  
				imgOut[iImg]=0;				
			}
			else{
				
				value=PyDoG[idxPyDoG].image[iImg];

				if(value > PyDoG[idxPyDoG-1].image[iImg-(1+imgC)] &&     
				   value > PyDoG[idxPyDoG-1].image[iImg-imgC] &&
				   value > PyDoG[idxPyDoG-1].image[iImg-(imgC-1)] &&
				   value > PyDoG[idxPyDoG-1].image[iImg-1] &&
				   value > PyDoG[idxPyDoG-1].image[iImg] &&
				   value > PyDoG[idxPyDoG-1].image[iImg+1] &&
				   value > PyDoG[idxPyDoG-1].image[iImg+(imgC-1)] &&
				   value > PyDoG[idxPyDoG-1].image[iImg+imgC] &&
				   value > PyDoG[idxPyDoG-1].image[iImg+(1+imgC)] &&
				   value > PyDoG[idxPyDoG].image[iImg-(1+imgC)] &&
				   value > PyDoG[idxPyDoG].image[iImg-imgC] &&
				   value > PyDoG[idxPyDoG].image[iImg-(imgC-1)] &&
				   value > PyDoG[idxPyDoG].image[iImg-1] &&
				   value > PyDoG[idxPyDoG].image[iImg+1] &&
				   value > PyDoG[idxPyDoG].image[iImg+(imgC-1)] &&
				   value > PyDoG[idxPyDoG].image[iImg+imgC] &&
				   value > PyDoG[idxPyDoG].image[iImg+(1+imgC)] &&
				   value > PyDoG[idxPyDoG+1].image[iImg-(1+imgC)] &&
				   value > PyDoG[idxPyDoG+1].image[iImg-imgC] &&
				   value > PyDoG[idxPyDoG+1].image[iImg-(imgC-1)] &&
				   value > PyDoG[idxPyDoG+1].image[iImg-1] &&
				   value > PyDoG[idxPyDoG+1].image[iImg] &&
				   value > PyDoG[idxPyDoG+1].image[iImg+1] &&
				   value > PyDoG[idxPyDoG+1].image[iImg+(imgC-1)] &&
				   value > PyDoG[idxPyDoG+1].image[iImg+imgC] &&
				   value > PyDoG[idxPyDoG+1].image[iImg+(1+imgC)]) {///Max
					imgOut[iImg]=1;
				}else if(value < PyDoG[idxPyDoG-1].image[iImg-(1+imgC)] &&     
				   value < PyDoG[idxPyDoG-1].image[iImg-imgC] &&
				   value < PyDoG[idxPyDoG-1].image[iImg-(imgC-1)] &&
				   value < PyDoG[idxPyDoG-1].image[iImg-1] &&
				   value < PyDoG[idxPyDoG-1].image[iImg] &&
				   value < PyDoG[idxPyDoG-1].image[iImg+1] &&
				   value < PyDoG[idxPyDoG-1].image[iImg+(imgC-1)] &&
				   value < PyDoG[idxPyDoG-1].image[iImg+imgC] &&
				   value < PyDoG[idxPyDoG-1].image[iImg+(1+imgC)] &&
				   value < PyDoG[idxPyDoG].image[iImg-(1+imgC)] &&
				   value < PyDoG[idxPyDoG].image[iImg-imgC] &&
				   value < PyDoG[idxPyDoG].image[iImg-(imgC-1)] &&
				   value < PyDoG[idxPyDoG].image[iImg-1] &&
				   value < PyDoG[idxPyDoG].image[iImg+1] &&
				   value < PyDoG[idxPyDoG].image[iImg+(imgC-1)] &&
				   value < PyDoG[idxPyDoG].image[iImg+imgC] &&
				   value < PyDoG[idxPyDoG].image[iImg+(1+imgC)] &&
				   value < PyDoG[idxPyDoG+1].image[iImg-(1+imgC)] &&
				   value < PyDoG[idxPyDoG+1].image[iImg-imgC] &&
				   value < PyDoG[idxPyDoG+1].image[iImg-(imgC-1)] &&
				   value < PyDoG[idxPyDoG+1].image[iImg-1] &&
				   value < PyDoG[idxPyDoG+1].image[iImg] &&
				   value < PyDoG[idxPyDoG+1].image[iImg+1] &&
				   value < PyDoG[idxPyDoG+1].image[iImg+(imgC-1)] &&
				   value < PyDoG[idxPyDoG+1].image[iImg+imgC] &&
				   value < PyDoG[idxPyDoG+1].image[iImg+(1+imgC)]){//Min
					imgOut[iImg]=1;
				} else
				{
					imgOut[iImg]=0;

				}
			
            }
		}
	}
	mM[idxmM].minMax=imgOut;
}


__global__ void RemoveOutlier(ArrayImage* PyDoG, MinMax * mM, int idxmM, int idxPyDoG, int imgR,int imgC ,float* auxOut)
{
	int tid= threadIdx.x;
	int bid= blockIdx.x;
	int bDim=blockDim.x;
	int gDim=gridDim.x;
	
		
	int iImg=0;
	int pxlThrd = ceil((double)(imgC*imgR)/(gDim*bDim)); ////////numero de veces que caben
														 ////////los hilos en la imagen.
	for(int i = 0; i <pxlThrd; ++i)///////////////////////////// Strike 
	{
		//////////////////////////////////////
		//////////////////////////////////////Calculo de indices
		iImg=(tid+(bDim*bid)) + (i*gDim*bDim); //// pixel en el que trabajara el hilo
		//////////////////////////////////////
		//////////////////////////////////////
		
		if(iImg < imgC*imgR){
			
			

			if(mM[idxmM].minMax[iImg]>0 && 	fabs(PyDoG[idxPyDoG].image[iImg])> 0.024)
			{
				
				float d, dxx, dyy, dxy, tr, det;
				d = PyDoG[idxPyDoG].image[iImg];
				dxx = PyDoG[idxPyDoG].image[iImg-imgC]+ PyDoG[idxPyDoG].image[iImg+imgC] - 2*d;
				dyy = PyDoG[idxPyDoG].image[iImg-1]+ PyDoG[idxPyDoG].image[iImg+1] - 2*d;
				dxy = (PyDoG[idxPyDoG].image[iImg-imgC-1] + PyDoG[idxPyDoG].image[iImg+1+imgC] - PyDoG[idxPyDoG].image[iImg+imgC-1] - PyDoG[idxPyDoG].image[iImg-imgC+1])/4.0;
				tr = dxx + dyy;
				det = dxx*dyy - dxy*dxy;
				/*
				if(det <= 0 )
					mM[idxmM].minMax[iImg]=0;
				else if( (tr*tr/det) < 12.1){
					mM[idxmM].minMax[iImg]=1;
				}else{
					mM[idxmM].minMax[iImg]=0;
				}*/

				if(det<0 || tr*tr/det > 7.2)
				{
					mM[idxmM].minMax[iImg]=0;
				}


			}else
			{
				mM[idxmM].minMax[iImg]=0;
			}

			auxOut[iImg]=mM[idxmM].minMax[iImg];
			

		}
	}
}



__global__ void OriMag(ArrayImage* PyDoG, int idxPyDoG, int imgR,int imgC , ArrayImage* Mag, ArrayImage* Ori, int idxMagOri, float* MagAux, float* OriAux) 
{
	int tid= threadIdx.x;
	int bid= blockIdx.x;
	int bDim=blockDim.x;
	int gDim=gridDim.x;
	float dx,dy;
			
	int iImg=0;
	int pxlThrd = ceil((double)(imgC*imgR)/(gDim*bDim)); ////////numero de veces que caben
														 ////////los hilos en la imagen.
	for(int i = 0; i <pxlThrd; ++i)///////////////////////////// Strike 
	{
		//////////////////////////////////////
		//////////////////////////////////////Calculo de indices
		iImg=(tid+(bDim*bid)) + (i*gDim*bDim); //// pixel en el que trabajara el hilo
		//////////////////////////////////////
		//////////////////////////////////////
		
		if(iImg < imgC*imgR){
			int condition=1/2+imgC*(floor((double)1/2));
			if (iImg-condition < 0  ||										///condicion arriba
				iImg+condition > imgC*imgR ||								///condicion abajo
				iImg%imgC < 1/2 ||										///condicion izquierda
				iImg%imgC > (imgC-1)-(1/2) )							///condicion derecha
			{                  
				OriAux[iImg]=0;
				MagAux[iImg]=0;

			}
			else{
				dx=PyDoG[idxPyDoG].image[iImg+1]-PyDoG[idxPyDoG].image[iImg-1];
				dy=PyDoG[idxPyDoG].image[iImg+imgC]-PyDoG[idxPyDoG].image[iImg-imgC];
				
				MagAux[iImg]=sqrt(dx*dx + dy*dy);

				OriAux[iImg]=atan2(dy,dx);
            }
		}
	}
	
	Mag[idxMagOri].image= MagAux;
	Ori[idxMagOri].image= OriAux;
}



__global__ void KeyPoints(ArrayImage * Mag, ArrayImage * Ori, MinMax * mM , int idxMOmM, keyPoint * KP, float sigma, int imgR,int imgC, int octava )
{
	int tid= threadIdx.x;
	int bid= blockIdx.x;
	int bDim=blockDim.x;
	int gDim=gridDim.x;
	float o = 0, val=0;
	int x=0, y=0, octv=-1;

	


	int iImg=0;
	int pxlThrd = ceil((double)(imgC*imgR)/(gDim*bDim)); ////////numero de veces que caben
														 ////////los hilos en la imagen.
	for(int i = 0; i <pxlThrd; ++i)///////////////////////////// Strike 
	{
		//////////////////////////////////////
		//////////////////////////////////////Calculo de indices
		iImg=(tid+(bDim*bid)) + (i*gDim*bDim); //// pixel en el que trabajara el hilo
		//////////////////////////////////////
		//////////////////////////////////////
		octv=-1;
		if(iImg < imgC*imgR ){

			if(mM[idxMOmM].minMax[iImg]>0 ){
				
					float histo[36]={0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0};
					octv=octava;
					x=iImg%imgC;
					y=iImg/imgC;
					
					int idxMO= (iImg-5)-(5*imgC);
					float exp_denom = 2.0 * sigma * sigma;
					float w;
					int bin;

					for (int i = -5; i < 6; ++i)
					{
						for (int j = -5; j < 6; ++j)
						{
							w = exp( -( i*i + j*j ) / exp_denom );
	  						bin =(Ori[idxMOmM].image[idxMO]<0)?round((double) (18*(6.283185307-Ori[idxMOmM].image[idxMO])/3.141592654)): round((double) (18*Ori[idxMOmM].image[idxMO]/3.141592654));
	  						histo[bin]+= w*Mag[idxMOmM].image[idxMO];
	  						++idxMO;
						}
						idxMO=idxMO+imgC-11;

					}



					int idxH=0;
					float valMaxH = histo[0];
					for (int i = 1; i < 36; ++i)
					{	
						
						if(histo[i]>valMaxH){
							idxH = i;
							valMaxH=histo[i]; 
							
						}
					}


					//printf("%f\n", valMaxH);

					int l = (idxH == 0)? 35:idxH-1;
					int r = (idxH+1)%36;

					float bin_;
					bin_= idxH + ((0.5*(histo[l]-histo[r]))/(histo[l]-(2*histo[idxH])+histo[r]));
					
							
				

					bin_= ( bin_ < 0 )? 36 + bin_ : ( bin_ >= 36 )? bin_ - 36 : bin_;
					
					o=((360*bin_)/36);//-3.141592654;
					val=valMaxH; 
        	}
        	else{
        		o=-1.0;
				x=-1;
				y=-1;
				octv=-1;


        	}
        	KP[iImg].orientacion=o;
		    KP[iImg].x=x;
		    KP[iImg].y=y;
		    KP[iImg].octv=octv;
		    KP[iImg].size=val;




		}
	}
}



__global__ void CountKeyPoint(MinMax * mM, int idxmM, int imgR, int imgC, int * numKeyP)
{
	int tid= threadIdx.x;
	int bDim=blockDim.x;
	
	
	__shared__ int num;
	int iImg=0;
	int pxlThrd = ceil((double)(imgC*imgR)/bDim); ////////numero de veces que caben
	if(tid==0) num=0;
	__syncthreads();
															
	for(int i = 0; i < pxlThrd; ++i)///////////////////////////// Strike
	{
		iImg= tid+(i*bDim);
		if(iImg < imgC*imgR && mM[idxmM].minMax[iImg]>0){
			atomicAdd(&num,1);
		}

	}

	numKeyP[0]=num;
}

void MaskGenerator(double sigma, int size,Mat mask){//Generate Gaussian Kernel
	Mat aux = getGaussianKernel(size,sigma,CV_32F);
	Mat aux_t;
	transpose(aux,aux_t);
	mask=aux*aux_t;
	

}

int ResizeImage(Mat image,vector<Mat>& images, int octvs){
	images.push_back(image);
	for(int i=0; i<octvs-1; ++i)
	{
		Mat aux = images[i];
		resize(aux,aux,Size(images[i].cols/2,images[i].rows/2));
		images.push_back(aux);
	}
	return 0;
}

int PyramidKDoG(vector<Mat> & PyKDoG, int octvs, int intvls){
	vector<double> sig;
	double sigma =sqrt(2.0f);
	vector<Mat> PyGauss;
	Mat resizeI;
	int size = 11;//size of gaussian mask
	Mat mask=Mat::ones(size,size,CV_32F);
	MaskGenerator(1,size,mask);
	PyGauss.push_back(mask);
	
	for(int i=1; i<intvls+3; ++i){	
		Mat aux=Mat::ones(size,size,CV_32F);
		double sigmaf=sqrt(pow(2.0,2.0/intvls)-1) * sigma;
		sigma= pow(2.0,1.0/ intvls ) * sigma;
		MaskGenerator(sigmaf,size,aux);
		PyGauss.push_back(aux);
	}
	
	//////////////////////////////
	/////////////////////////////////////////////////////////
	/////////////////////////////////////////////////////////////////////Resta de Gausianas
	for(int i=0; i<intvls+2; ++i){
		Mat aux=Mat::ones(size,size,CV_32F);
		subtract(PyGauss[i+1],PyGauss[i],aux);
		PyKDoG.push_back(aux);
	}
	///////////////////////////////////////////////////////////////////////////////////////


	
	return 0;
}

int foundIndexesMaxMin(float* minMax,vector<int*> & idxMinMax, int count )
{
	vector<int> idxmM;
	for (int c = 0; c <  count; ++c)
	{
		
		if (minMax[c]==0.0)
		{
			idxmM.push_back(c);
			//cout<<c<<endl;
		}
	}
	idxMinMax.push_back(idxmM.data());
	

	return 0;
}

float SiftFeatures(Mat Image, vector<Mat> PyDoG,Mat I){
	const int intvls = 3;
	int octvs;
	//hipError_t e;
	octvs = log( min( Image.rows, Image.cols ) ) / log(2) - 2;
	vector<Mat> PyKDoG;
	vector<Mat> images;
	ArrayImage * pyDoG;
	MinMax * minMax;
	int mMidx=1;
	int idxPyDoG=0;



	hipEvent_t start, stop;
 	hipEventCreate(&start);
 	hipEventCreate(&stop);


 	hipEventRecord(start, 0);

	PyramidKDoG( PyKDoG,octvs,intvls);
	ResizeImage(Image,images,octvs);
	
	////////////////////////////////////////////////////////////////////////////////////////
	/////////////////////////////////////////////////////////////////////Reservo Memoria GPU

	hipMalloc(&pyDoG,sizeof(ArrayImage)*images.size()*PyKDoG.size());
	hipMalloc(&minMax,sizeof(MinMax)*intvls*images.size());
	//cout<<hipGetErrorString(e)<<" hipMalloc"<<endl;
	
	for (int i = 0; i < images.size() ; ++i)
	{
		
		float * img_D;
		int sizeImage = images[i].rows*images[i].cols;
		
		////////////////////////////////////////////////////////////////////////////////////////
		/////////////////////////////////////////////////////////////////////Reservo Memoria GPU
		
		hipMalloc(&img_D,sizeof(float)*sizeImage);///imagenes
		//cout<<hipGetErrorString(e)<<" hipMalloc"<<endl;
	
		////////////////////////////////////////////////////////////////////////////////////////
		/////////////////////////////////////////////////////////////////////Copio Memoria GPU

		hipMemcpy(img_D,images[i].ptr<float>(),sizeof(float)*sizeImage,hipMemcpyHostToDevice);
		//cout<<hipGetErrorString(e)<<" cudaMemCopyHD"<<endl;

		int imgBlocks= ceil((double) images[i].cols/BW);
		
		////////////////////////////////////////////////////////////////////////////////////////
		///////////////////////////////////////////////////////Convolucion de mascara con imagen
		/////////////////////////////////////////////////////////////////////Una Octava or ciclo
		for (int m = 0; m < PyKDoG.size(); ++m){
			float * pkDoG_D;
			float * out_D;
			//float * out= new float[sizeImage];
			int sizeMask=PyKDoG[m].rows*PyKDoG[m].cols;

			////////////////////////////////////////////////////////////////////////////////////////
			/////////////////////////////////////////////////////////////////////Reservo Memoria GPU
			hipMalloc(&pkDoG_D,sizeof(float)*sizeMask);//mascaras
			//cout<<hipGetErrorString(e)<<" cudaMalloc________Mask "<<endl;
			hipMalloc(&out_D,sizeof(float)*sizeImage);
			//cout<<hipGetErrorString(e)<<" cudaMalloc________Mask"<<endl;
			////////////////////////////////////////////////////////////////////////////////////////
			/////////////////////////////////////////////////////////////////////Copio Memoria GPU

			hipMemcpy(pkDoG_D,PyKDoG[m].ptr<float>(),sizeof(float)*sizeMask,hipMemcpyHostToDevice);
			//cout<<hipGetErrorString(e)<<" cudaMemCopyHD________Mask"<<endl;

			////////////////////////////////////////////////////////////////////////////////////////
			/////////////////////////////////////////////////////////////////////Lanzo Kernel
			
			Convolution<<<imgBlocks,1024>>>(img_D,pkDoG_D,pyDoG,PyKDoG[m].rows,PyKDoG[m].cols,images[i].rows,images[i].cols,out_D,idxPyDoG);
			//hipDeviceSynchronize();
			++idxPyDoG;
			hipFree(pkDoG_D); 
			
			
			//hipMemcpy(out,out_D,sizeof(float)*sizeImage,hipMemcpyDeviceToHost);
			//cout<<hipGetErrorString(e)<<" cudaMemCopyDH________Mask"<<endl;

			//Mat image_out(images[i].rows,images[i].cols,CV_32F,out);
			

			//cout<<image_out<<endl;
			//imshow("PyDoG",image_out);
    		//waitKey(0);
    		//destroyAllWindows();
			
			//delete(out);
			//hipFree(out_D);
		}
		hipFree(img_D);
		////////////////////////////////////////////////////////////////////////////////////////
		////////////////////////////////////////////////////////////////////////////////////////
		////////////////////////////////////////////////////////////////////////////////////////
	}

	int maskC =PyKDoG[0].cols;
	int idxmM=0;
	for (int i = 0; i <images.size() ; ++i)
	{
		int sizeImage = images[i].rows*images[i].cols;
		int imgBlocks= ceil((double) images[i].cols/BW);
		////////////////////////////////////////////////////////////////////////////////////////
		//////////////////////////////////////////////////////////////////////Busqueda de MinMax
		/////////////////////////////////////////////////////////////////////Una Octava or ciclo
		
		int m=0;
		for(m = mMidx; m < mMidx+intvls; ++m){
			
			float * out_D;
			//float * out = new float[sizeImage];
			////////////////////////////////////////////////////////////////////////////////////////
			/////////////////////////////////////////////////////////////////////Reservo Memoria GPU
			hipMalloc(&out_D,sizeof(float)*sizeImage);
			//cout<<hipGetErrorString(e)<<" cudaMalloc________Mask"<<endl;
						
			////////////////////////////////////////////////////////////////////////////////////////
			/////////////////////////////////////////////////////////////////////Lanzo Kernel
			///////entrega ya los puntos descartanbdo los de bajo contraste
			LocateMaxMin<<<imgBlocks,1024>>>(pyDoG,m,out_D,minMax,maskC,images[i].rows,images[i].cols,idxmM);
			++idxmM;
			//hipDeviceSynchronize();

			//hipMemcpy(out,out_D,sizeof(float)*sizeImage,hipMemcpyDeviceToHost);
			//cout<<hipGetErrorString(e)<<" cudaMemCopyDH________Mask"<<endl;

			//Mat image_out(images[i].rows,images[i].cols,CV_32F,out);
			
			//imshow("MinMax",image_out);

    		//waitKey(0);
    		//destroyAllWindows();
			
			//delete(out);
			
		}
		mMidx=m+2;
		
		////////////////////////////////////////////////////////////////////////////////////////
		////////////////////////////////////////////////////////////////////////////////////////
		////////////////////////////////////////////////////////////////////////////////////////
	}

	///////////////////////////////////////////////////////////////////////////////////////////////////
	///////////////////////////////////////////////////////////////////////////////////////////////////
	/////////////////////////////////////////////////////////////////////Remover outliers



	idxPyDoG=1, idxmM=0;
	
	for(int i = 0; i< images.size(); ++i )
	{	float* out_D;
		int sizeImage = images[i].rows*images[i].cols;
		int imgBlocks= ceil((double) images[i].cols/BW);
		hipMalloc(&out_D,sizeof(float)*sizeImage);
		//float * out = new float[sizeImage];

		for (int j = 0; j < intvls; ++j)
		{
			RemoveOutlier<<<imgBlocks,1024>>>(pyDoG,minMax,idxmM,idxPyDoG, images[i].rows,images[i].cols,out_D);
			//hipMemcpy(out,out_D,sizeof(float)*sizeImage,hipMemcpyDeviceToHost);
			

			//Mat image_out(images[i].rows,images[i].cols,CV_32F,out);
			
			//imshow("MinMax Filtrados",image_out);
    		//waitKey(0);
    		//destroyAllWindows();
    		
    		
			++idxmM;
			++idxPyDoG;
		}
		idxPyDoG+=2;

		//delete(out);
		hipFree(out_D);
	}
	///////////////////////////////////////////////////////////////////////////////////////////////////
	///////////////////////////////////////////////////////////////////////////////////////////////////
	/////////////////////////////////////////////////////////////////////Calculo de Orientaciones y magnitud en DoG
	

	ArrayImage * Mag;
	ArrayImage * Ori;
	

	hipMalloc(&Mag,sizeof(ArrayImage)*intvls*images.size());
	hipMalloc(&Ori,sizeof(ArrayImage)*intvls*images.size());

	idxPyDoG=1;
	int idxMagOri=0;
	for(int i = 0; i< images.size(); ++i )
	{	
		float * MagAux;
		float * OriAux;
		int sizeImage = images[i].rows*images[i].cols;
		int imgBlocks= ceil((double) images[i].cols/BW);
		hipMalloc(&MagAux,sizeof(float)*sizeImage);
		hipMalloc(&OriAux,sizeof(float)*sizeImage);
		//float * out = new float[sizeImage];

		for (int j = 0; j < intvls; ++j)
		{
			OriMag<<<imgBlocks,1024>>>(pyDoG,idxPyDoG, images[i].rows,images[i].cols,Mag,Ori,idxMagOri,MagAux,OriAux);
			//hipMemcpy(out,OriAux,sizeof(float)*sizeImage,hipMemcpyDeviceToHost);
			

			//Mat image_out(images[i].rows,images[i].cols,CV_32F,out);
			
			//imshow("tesuto",image_out);
    		//waitKey(0);
    		//destroyAllWindows();
			
			++idxMagOri;
			++idxPyDoG;
		}
		idxPyDoG+=2;



		//delete(out);
		//hipFree(MagAux);
		//hipFree(OriAux);
	}

	///////////////////////////////////////////////////////////////////////////////////////////////////
	///////////////////////////////////////////////////////////////////////////////////////////////////
	/////////////////////////////////////////////////////////////////////Obtener orientacion de keypoints

	//vector<KeyPoint> KPoints;
	
	
	idxmM=0;
	for(int i = 0; i< images.size(); ++i )
	{
		float sigma=sqrt(2.0f);
		int imgBlocks= ceil((double) images[i].cols/BW);
		keyPoint * KP;
		//keyPoint * KP_host = new keyPoint[images[i].rows*images[i].cols];
		
		hipMalloc(&KP,sizeof(keyPoint)*images[i].rows*images[i].cols); 
		for (int j = 0; j < intvls; ++j)
		{
			KeyPoints<<<imgBlocks,1024>>>(Mag, Ori,  minMax , idxmM,  KP, sigma, images[i].rows,images[i].cols, i );
			//hipMemcpy(KP_host,KP,sizeof(keyPoint)*images[i].rows*images[i].cols,hipMemcpyDeviceToHost);

			sigma= pow(2.0,1.0/ intvls ) * sigma;
			++idxmM;
			/*
			
			
			for(int k=0; k<(images[i].rows*images[i].cols); ++k){
				

				
				if( !(KP_host[k].octv <0) ){
					//cout<<idxmM<<endl;
					if (i>0)
					{
						KP_host[k].x*=pow(2,i);
						KP_host[k].y*=pow(2,i);
					}
					KeyPoint aux(KP_host[k].x,KP_host[k].y,KP_host[k].size,KP_host[k].orientacion ,0,KP_host[k].octv);
					//cout<<KP_host[k].size<<endl;
					KPoints.push_back(aux);
				}
			}*/
		}
		//delete(KP_host);
		hipFree(KP);
	}

	hipEventRecord(stop, 0);
 	hipEventSynchronize(stop);
 
 	float elapsedTime;
 	hipEventElapsedTime(&elapsedTime,start, stop);
 	cout<< "Tiempo total "<<elapsedTime << " en milseg"<<endl;

 	hipEventDestroy(start);
 	hipEventDestroy(stop);


 	//cout<<KPoints.size()<<endl;
 	
 	
	Mat out;
	//drawKeypoints(I,KPoints,out);
	//imshow("Puntos Caracteristicos SIFT",out);
    //waitKey(0);
    //destroyAllWindows();


    /*Ptr<DescriptorExtractor> featureExtractor = DescriptorExtractor::create("SIFT");
    Mat descriptors;
  	featureExtractor->compute(I, KPoints, descriptors);

  	
  	Mat outputImage;
  	Scalar keypointColor = Scalar(255, 0, 0);   
  	drawKeypoints(I, KPoints, outputImage, keypointColor, DrawMatchesFlags::DEFAULT);
	

	imshow("test",outputImage);
	waitKey(0);
	destroyAllWindows();*/


	hipFree(Ori);
	hipFree(Mag);
	hipFree(pyDoG);
	hipFree(minMax);

	return elapsedTime;
}




